
#include "hip/hip_runtime.h"


#include <stdio.h>


#include <omp.h>
#include <string>
#include <fstream>
#include <vector>
#include <iomanip>
#include <iostream>
#include <sstream>

using namespace std;

struct GpuStruct
{
	char *pav;
	int kiekis;
	double kaina;
};

class Struct
{
	string pav;
	int kiekis;
	double kaina;
	GpuStruct gpuStruct;
public:
	Struct(string input);
	~Struct(){hipFree(gpuStruct.pav);}
	GpuStruct GetDev(){return gpuStruct;}
	string Print();
};

Struct::Struct(string input)
{
	int start, end;
	start = 0;
	end = input.find(' ');
	pav = input.substr(0, end).c_str();
	start = end + 1;
	end = input.find(' ', start);
	kiekis = stoi(input.substr(start, end - start));
	start = end + 1;
	kaina = stod(input.substr(start));
	gpuStruct.kaina = kaina;
	gpuStruct.kiekis = kiekis;
	hipMalloc(&gpuStruct.pav, pav.size() + 1);
	hipMemcpy(gpuStruct.pav, pav.c_str(), pav.size() + 1, hipMemcpyHostToDevice);
}

string Struct::Print()
{
	stringstream ss;
	ss << setw(15) << pav << setw(7) << kiekis << setw(20) << kaina;
	return ss.str();
}

vector<vector<Struct>> ReadStuff(string file);
vector<string> ReadLines(string file);

string Titles();
string Print(int nr, Struct &s);
void syncOut(vector<vector<Struct>>&);

__global__ void DevPrint(GpuStruct *data, int* starts);

int main()
{
	auto input = ReadStuff("LapunasD.txt");
	int count = 0;
	for(auto &vec : input)
		count += vec.size();
	cout << "\nsinchroninis isvedimas\n\n";
	syncOut(input);
	cout << "\nasinchroninis isvedimas\n\n";
	cout << setw(10) << "Procesas" << setw(3) << "Nr" << Titles() << "\n\n";
	
	vector<int> starts;
	vector<GpuStruct> localStructs;
	
	int put = 0;
	for(auto &vec : input)
	{
		starts.push_back(put);
		for(auto &s : vec)
		{
			localStructs.push_back(s.GetDev());
			put++;
		}
	}
	starts.push_back(put);
	int *startsdev;
	hipMalloc(&startsdev, sizeof(int) * starts.size());
	hipMemcpy(startsdev, &starts[0], sizeof(int) * starts.size(), hipMemcpyHostToDevice);
	GpuStruct *arr;
	hipMalloc(&arr, sizeof(GpuStruct) * count);
	hipMemcpy(arr, &localStructs[0], sizeof(GpuStruct) * count, hipMemcpyHostToDevice);
	DevPrint<<<1, input.size()>>>(arr, startsdev);
	hipDeviceSynchronize();
	system("pause");
	hipFree(arr);
	hipFree(startsdev);
	return 0;
}

vector<vector<Struct>> ReadStuff(string file)
{
	auto lines = ReadLines(file);
	vector<vector<Struct>> ret;
	vector<Struct> tmp;
	for(int i = 0; i < lines.size(); i++)
	{
		if(lines[i] == "")
		{
			ret.push_back(move(tmp));
		}
		else
		{
			tmp.emplace_back(lines[i]);
		}
	}
	return ret;
}

vector<string> ReadLines(string file)
{
	vector<string> ret;
	ifstream duom(file);
	while(!duom.eof())
	{
		string line;
		getline(duom, line);
		ret.push_back(line);
	}
	return ret;
}

string Titles()
{
	stringstream ss;
	ss << setw(15) << "Pavadiniams" << setw(7) << "Kiekis" << setw(20) << "Kaina";
	return ss.str();
}

void syncOut(vector<vector<Struct>> &data)
{
	cout << setw(3) << "Nr" << Titles() << endl << endl;
	for(int i = 0; i < data.size(); i++)
	{
		auto &vec = data[i];
		cout << "Procesas_" << i << endl;
		for(int j = 0; j < vec.size(); j++)
		{
			cout << Print(j, vec[j]) << endl;
		}
	}
}

string Print(int nr, Struct &s)
{
	stringstream ss;
	ss << setw(3) << nr << s.Print();
	return ss.str();
}

__global__ void DevPrint(GpuStruct *data, int *starts)
{
	int id = threadIdx.x;
	GpuStruct *d = data + starts[id];
	int count = starts[id+1] - starts[id];
	for(int i = 0; i < count; i++)
	{
		printf("Procesas_%i %2i %14s %6i %19f\n", id, i, d[i].pav, d[i].kiekis, d[i].kaina);
	}
}